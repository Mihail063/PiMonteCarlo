
#include "hip/hip_runtime.h"

#include <iostream>
#include <time.h>
#include <stdio.h>
#include<hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE  16         
#define N           1024     
#define MAX         1000
__global__ void piCalk(double*a, double*b, int n, int* circle_points)
{
    int   bx = blockIdx.x;    
    int   by = blockIdx.y;
    int   tx = threadIdx.x;     
    int   ty = threadIdx.y;   
    int   ia = BLOCK_SIZE * by + ty;  
    int   ib = BLOCK_SIZE * bx + tx;
    double V = 0.0;
    for (int k = 0 ; k < n; k++)
    {
        V = pow(a[ia + k], 2) + pow(b[ib + k], 2);
        if (V < 1)
            *circle_points++;
    }
}

__global__ void random(unsigned int seed, int* result) {

    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    *result = hiprand(&state) % MAX;
}

int randGPU()
{
    int* gpu_x;
    hipMalloc((void**)&gpu_x, sizeof(int));

    random <<<1, 1 >>> (time(NULL), gpu_x);
    int x;
    hipMemcpy(&x, gpu_x, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpu_x);
    return x;
}
int main()
{
    int numBytes = N * sizeof(float);

    double* X = new double[N];
    double* Y = new double[N];
    int* circle_points = new int(0);
    double* xdev = NULL;
    double* ydev = NULL;
    for (int i = 0; i < N; i++)
    {   
        double xi = double(randGPU()) / MAX;
        X[i] = xi;
        double yi = double(randGPU()) / MAX;
        Y[i] = yi;
    }
    hipMalloc((void**)&xdev, numBytes);
    hipMalloc((void**)&ydev, numBytes);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMemcpy(xdev, X, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(ydev, Y, numBytes, hipMemcpyHostToDevice);
    for (int i = 0; i < N; i++)
    {
        printf("%d., %d.\n", X[i], Y[i]);
    }
    piCalk <<<blocks, threads >>> (xdev, ydev, N, circle_points);
    double pi = double(4 * *circle_points) / N;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing by the GPU: %.2f millseconds\n %d", gpuTime, pi);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(xdev);
    hipFree(ydev);

    delete X;
    delete Y;
    return 0;
}
